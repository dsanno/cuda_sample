#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MATRIX_SIZE 1024
#define BLOCK_SIZE 32
#define SHARED_BLOCK_SIZE 128

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int main(int argc, char** argv){
	int matrixSize = sizeof(int)* MATRIX_SIZE * MATRIX_SIZE;

	int* hMatrixA;
	int* hMatrixB;
	int* hMatrixC;
	hMatrixA = (int*)malloc(matrixSize);
	hMatrixB = (int*)malloc(matrixSize);

	/* Matrix�̏����l�ݒ� */
	int col_idx, row_idx;
	for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++){
		for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++){
			hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % 1024;
			hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % 1024;
		}
	}

	/* �f�o�C�X���̕ϐ��ݒ� */
	int* dMatrixA;
	int* dMatrixB;
	int* dMatrixC;

	/* �f�o�C�X�������̈�̊m�� */
	hipMalloc((void**)&dMatrixA, matrixSize);
	hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&dMatrixB, matrixSize);
	hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&dMatrixC, matrixSize);

	/* �u���b�N�T�C�Y�ƃO���b�h�T�C�Y�̐ݒ� */
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(MATRIX_SIZE / BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);

	hipError_t error;
	/* �^�C�}�[���쐬���Čv���J�n */
	hipEvent_t start;
	error = hipEventCreate(&start);
	if (error != hipSuccess) {
		printf("failed to craete start event");
		exit(EXIT_FAILURE);
	}
	hipEvent_t stop;
	error = hipEventCreate(&stop);
	if (error != hipSuccess) {
		printf("failed to crete stop event");
		exit(EXIT_FAILURE);
	}

	error = hipEventRecord(start, NULL);
	if (error != hipSuccess) {
		printf("failed to record start event");
		exit(EXIT_FAILURE);
	}

	/* �J�[�l���̋N�� */
	matrixMul << <grid, block >> >(dMatrixA, dMatrixB, dMatrixC);
	hipDeviceSynchronize();

	/* ���ʂ̗̈�m�ۂƃf�o�C�X������̃������]�� */
	hMatrixC = (int*)malloc(matrixSize);
	hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost);

	error = hipEventRecord(stop, NULL);
	if (error != hipSuccess) {
		printf("failed to record stop event");
		exit(EXIT_FAILURE);
	}

	error = hipEventSynchronize(stop);
	if (error != hipSuccess) {
		printf("failed to synchronize");
		exit(EXIT_FAILURE);
	}

	/* �^�C�}�[���~ */
	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);
	if (error != hipSuccess) {
		printf("failed to get elapsed time");
		exit(EXIT_FAILURE);
	}
	printf("Processing time: %f (msec)\n", msecTotal);
	printf("%d, %d\n", hMatrixC[0], hMatrixC[MATRIX_SIZE * MATRIX_SIZE - 1]);

	int row = 235;
	int col = 739;
	int target = 0;
	for (int i = 0; i < MATRIX_SIZE; i++) {
		target += hMatrixA[row * MATRIX_SIZE + i] * hMatrixB[i * MATRIX_SIZE + col];
	}
	printf("%d, %d\n", target, hMatrixC[row * MATRIX_SIZE + col]);

		/* �z�X�g�E�f�o�C�X�������̊J�� */
	free(hMatrixA);
	free(hMatrixB);
	free(hMatrixC);
	hipFree(dMatrixA);
	hipFree(dMatrixB);
	hipFree(dMatrixC);

	/* �I������ */
	hipDeviceReset();
}

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC){
	int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
	int scan_idx, scan_base;
	int target = 0;

	__shared__ int a[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ int b[BLOCK_SIZE * BLOCK_SIZE];

	for (scan_base = 0; scan_base < MATRIX_SIZE; scan_base += BLOCK_SIZE) {
		/* shared memory �ɃR�s�[ */
		for (scan_idx = 0; scan_idx < BLOCK_SIZE; scan_idx += BLOCK_SIZE) {
			a[threadIdx.y * BLOCK_SIZE + scan_idx + threadIdx.x] = inMatrixA[row_idx * MATRIX_SIZE + scan_base + scan_idx + threadIdx.x];
			b[(scan_idx + threadIdx.y) * BLOCK_SIZE + threadIdx.x] = inMatrixB[(scan_base + scan_idx + threadIdx.y) * MATRIX_SIZE + col_idx];
		}
		__syncthreads();
		/* �s��̉��Z���s�� */
		for (scan_idx = 0; scan_idx < BLOCK_SIZE; scan_idx++) {
			target += a[threadIdx.y * BLOCK_SIZE + scan_idx] * b[scan_idx * BLOCK_SIZE + threadIdx.x];
		}
		__syncthreads();
	}
	inMatrixC[row_idx * MATRIX_SIZE + col_idx] = target;
}
