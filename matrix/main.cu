#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MATRIX_SIZE 1024
#define BLOCK_SIZE 32

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int main(int argc, char** argv){
	int matrixSize = sizeof(int)* MATRIX_SIZE * MATRIX_SIZE;

	int* hMatrixA;
	int* hMatrixB;
	int* hMatrixC;
	hMatrixA = (int*)malloc(matrixSize);
	hMatrixB = (int*)malloc(matrixSize);

	/* Matrix�̏����l�ݒ� */
	int col_idx, row_idx;
	for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++){
		for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++){
			hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % 1024;
			hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % 1024;
		}
	}

	/* �f�o�C�X���̕ϐ��ݒ� */
	int* dMatrixA;
	int* dMatrixB;
	int* dMatrixC;

	/* �f�o�C�X�������̈�̊m�� */
	hipMalloc((void**)&dMatrixA, matrixSize);
	hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&dMatrixB, matrixSize);
	hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&dMatrixC, matrixSize);

	/* �u���b�N�T�C�Y�ƃO���b�h�T�C�Y�̐ݒ� */
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(MATRIX_SIZE / BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);

	hipError_t error;
	/* �^�C�}�[���쐬���Čv���J�n */
	hipEvent_t start;
	error = hipEventCreate(&start);
	if (error != hipSuccess) {
		printf("failed to craete start event");
		exit(EXIT_FAILURE);
	}
	hipEvent_t stop;
	error = hipEventCreate(&stop);
	if (error != hipSuccess) {
		printf("failed to crete stop event");
		exit(EXIT_FAILURE);
	}

	error = hipEventRecord(start, NULL);
	if (error != hipSuccess) {
		printf("failed to record start event");
		exit(EXIT_FAILURE);
	}

	/* �J�[�l���̋N�� */
	matrixMul << <grid, block >> >(dMatrixA, dMatrixB, dMatrixC);
	hipDeviceSynchronize();

	/* ���ʂ̗̈�m�ۂƃf�o�C�X������̃������]�� */
	hMatrixC = (int*)malloc(matrixSize);
	hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost);

	error = hipEventRecord(stop, NULL);
	if (error != hipSuccess) {
		printf("failed to record stop event");
		exit(EXIT_FAILURE);
	}

	error = hipEventSynchronize(stop);
	if (error != hipSuccess) {
		printf("failed to synchronize");
		exit(EXIT_FAILURE);
	}

	/* �^�C�}�[���~ */
	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);
	if (error != hipSuccess) {
		printf("failed to get elapsed time");
		exit(EXIT_FAILURE);
	}
	printf("Processing time: %f (msec)\n", msecTotal);
	printf("%d, %d\n", hMatrixC[0], hMatrixC[MATRIX_SIZE * MATRIX_SIZE - 1]);

	int row = 235;
	int col = 739;
	int target = 0;
	for (int i = 0; i < MATRIX_SIZE; i++) {
		target += hMatrixA[row * MATRIX_SIZE + i] * hMatrixB[i * MATRIX_SIZE + col];
	}
	printf("%d, %d\n", target, hMatrixC[row * MATRIX_SIZE + col]);

	/* �z�X�g�E�f�o�C�X�������̊J�� */
	free(hMatrixA);
	free(hMatrixB);
	free(hMatrixC);
	hipFree(dMatrixA);
	hipFree(dMatrixB);
	hipFree(dMatrixC);

	/* �I������ */
	hipDeviceReset();
}

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC){
	int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
	int scan_idx;
	int target = 0;

	/*�s��̉��Z���s��*/
	for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
		target += inMatrixA[row_idx * MATRIX_SIZE + scan_idx] * inMatrixB[scan_idx * MATRIX_SIZE + col_idx];
	}
	
	inMatrixC[row_idx * MATRIX_SIZE + col_idx] = target;
}
