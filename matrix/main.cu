#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MATRIX_SIZE 1024
#define BLOCK_SIZE 32

typedef unsigned int uint;

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int main(int argc, char** argv){
	uint matrixSize = sizeof(uint)* MATRIX_SIZE * MATRIX_SIZE;

	int* hMatrixA;
	int* hMatrixB;
	int* hMatrixC;
	hMatrixA = (int*)malloc(matrixSize);
	hMatrixB = (int*)malloc(matrixSize);

	/* Matrix�̏����l�ݒ� */
	uint col_idx, row_idx;
	for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++){
		for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++){
			hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
			hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
		}
	}

	/* �f�o�C�X���̕ϐ��ݒ� */
	int* dMatrixA;
	int* dMatrixB;
	int* dMatrixC;

	/* �f�o�C�X�������̈�̊m�� */
	hipMalloc((void**)&dMatrixA, matrixSize);
	hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&dMatrixB, matrixSize);
	hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&dMatrixC, matrixSize);

	/* �u���b�N�T�C�Y�ƃO���b�h�T�C�Y�̐ݒ� */
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(MATRIX_SIZE / BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);

	hipError_t error;
	/* �^�C�}�[���쐬���Čv���J�n */
	hipEvent_t start;
	error = hipEventCreate(&start);
	if (error != hipSuccess) {
		printf("failed to craete start event");
		exit(EXIT_FAILURE);
	}
	hipEvent_t stop;
	error = hipEventCreate(&stop);
	if (error != hipSuccess) {
		printf("failed to crete stop event");
		exit(EXIT_FAILURE);
	}

	error = hipEventRecord(start, NULL);
	if (error != hipSuccess) {
		printf("failed to record start event");
		exit(EXIT_FAILURE);
	}

	/* �J�[�l���̋N�� */
	matrixMul << <grid, block >> >(dMatrixA, dMatrixB, dMatrixC);
	hipDeviceSynchronize();

	/* ���ʂ̗̈�m�ۂƃf�o�C�X������̃������]�� */
	hMatrixC = (int*)malloc(matrixSize);
	hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost);

	error = hipEventRecord(stop, NULL);
	if (error != hipSuccess) {
		printf("failed to record stop event");
		exit(EXIT_FAILURE);
	}

	error = hipEventSynchronize(stop);
	if (error != hipSuccess) {
		printf("failed to synchronize");
		exit(EXIT_FAILURE);
	}

	/* �^�C�}�[���~ */
	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);
	if (error != hipSuccess) {
		printf("failed to get elapsed time");
		exit(EXIT_FAILURE);
	}
	printf("Processing time: %f (msec)\n", msecTotal);
	printf("%d, %d\n", hMatrixC[0], hMatrixC[MATRIX_SIZE * MATRIX_SIZE - 1]);

	/* �z�X�g�E�f�o�C�X�������̊J�� */
	free(hMatrixA);
	free(hMatrixB);
	free(hMatrixC);
	hipFree(dMatrixA);
	hipFree(dMatrixB);
	hipFree(dMatrixC);

	/* �I������ */
	hipDeviceReset();
}

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC){
	uint col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint row_idx = blockIdx.y * blockDim.y + threadIdx.y;
	uint scan_idx;
	uint target = 0;

	/*�s��̉��Z���s��*/
	for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
		target += inMatrixA[col_idx * MATRIX_SIZE + scan_idx] * inMatrixB[scan_idx * MATRIX_SIZE + row_idx];
	}
	
	inMatrixC[col_idx * MATRIX_SIZE + row_idx] = target;
}
